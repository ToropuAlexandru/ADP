#include "hip/hip_runtime.h"
#include "imgproc.cuh"

int main(int argc, char** argv)
{
	/*
	
	CvCapture* capture = cvCaptureFromCAM(CV_CAP_ANY);
	if (!capture) {
		fprintf(stderr, "ERROR: capture is NULL \n");
		getchar();
		return -1;
	}
	// Create a window in which the captured images will be presented
	cvNamedWindow("mywindow", CV_WINDOW_AUTOSIZE);
	// Show the image captured from the camera in the window and repeat
	while (1) {
		// Get one frame
		IplImage* frame = cvQueryFrame(capture);
		if (!frame) {
			fprintf(stderr, "ERROR: frame is null...\n");
			getchar();
			break;
		}
		cvShowImage("mywindow", frame);
		// Do not release the frame!

		if ((cvWaitKey(10) & 255) == 's') {
			CvSize size = cvGetSize(frame);
			IplImage* img = cvCreateImage(size, IPL_DEPTH_16S, 1);
			img = frame;
			cvSaveImage("my.png", &img);
		}
		if (cvWaitKey(10) < 0)
			cvSaveImage("my.png", frame);
	}
	// Release the capture device housekeeping
	cvReleaseCapture(&capture);
	cvDestroyWindow("mywindow");
	*/

	namedWindow("src", WINDOW_AUTOSIZE);
	namedWindow("src", WINDOW_AUTOSIZE);
	string filename = "lena_rgb.png";
	Mat src, dst;
	src = imread(filename, CV_LOAD_IMAGE_GRAYSCALE);
	src.convertTo(src, CV_8UC1);
	clock_t start, stop;
	start = clock();
	medianGPU(src, dst);
	imwrite("lena_median.jpg", dst);

	stop = clock();
	cout << "All cost time is " << static_cast<double>(stop - start) * 1000 / CLOCKS_PER_SEC << " ms" << endl;
	dst = dst.rowRange(1, dst.rows - 1).colRange(1, dst.cols - 1);

	imshow("src", src);
	imshow("dst", dst);
	waitKey(0);
	return 0;
}
