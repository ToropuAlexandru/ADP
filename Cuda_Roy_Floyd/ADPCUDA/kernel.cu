#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

__global__
void RoyFloyd(int* mat, int k, int N) {

	int i = blockIdx.x;
	int j = threadIdx.x;
	mat[i*N + j] = (mat[i*N + k] + mat[k*N + j]) < mat[i*N + j] ?
					(mat[i*N + k] + mat[k*N + j]) : 
					mat[i*N + j];

	
}


int main(int argc, char **argv)
{
	
	int thread_per_block = 512, i, j;
	size_t N = 700;
	int* cuda_mat;
	int size = sizeof(int)* N * N;
	time_t  time_start, time_end;
	

	srand(time(NULL));
	int *mat = (int*)malloc(sizeof(int)*N*N);
	for (int i = 0; i < N*N; i++)
		mat[i] = rand() % 500;
	for (int i = 0; i < N; i++)
		mat[i*N + i] = 0;

	time(&time_start);
	int *result = (int*)malloc(sizeof(int)*N*N);
	memcpy(result, mat, sizeof(int)*N*N);

	hipMalloc((void**)&cuda_mat, size);
	hipMemcpy(cuda_mat, mat, size, hipMemcpyHostToDevice);
	int num_block = ceil((float)(N*N / (thread_per_block)));
	for (int k = 0; k < N; ++k) {
		RoyFloyd << <num_block, (thread_per_block) >> >(cuda_mat, k, N);
	}
	hipMemcpy(mat, cuda_mat, size, hipMemcpyDeviceToHost);
	hipFree(cuda_mat);
	time(&time_end);

	//printf("%lld\n", (long long)(time_end - time_start));
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
			printf("dist[%d][%d] = %d \n", i, j, result[i*N + j]);
	}

}